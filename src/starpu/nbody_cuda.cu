#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2024  University of Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>
#include "../include/body.h"

static __global__ void bodyForce(Pos *p, Vel *v, int n, int offset)
{
	int initialIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = initialIndex; i < n; i += stride)
	{
		float Fx = 0.0f;
		float Fy = 0.0f;
		float Fz = 0.0f;

		for (int j = 0; j < n; j++)
		{
			float dx = p[j].x - p[i + offset].x;
			float dy = p[j].y - p[i + offset].y;
			float dz = p[j].z - p[i + offset].z;
			float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
			float invDist = rsqrtf(distSqr);
			float invDist3 = invDist * invDist * invDist;

			Fx += dx * invDist3;
			Fy += dy * invDist3;
			Fz += dz * invDist3;
		}

		v[i].vx += dt * Fx;
		v[i].vy += dt * Fy;
		v[i].vz += dt * Fz;
	}
}

static __global__ void integratePositions(Pos *p, Vel *v, int n)
{
	int initialIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = initialIndex; i < n; i += stride)
	{ // integrate position
		p[i].x += v[i].vx * dt;
		p[i].y += v[i].vy * dt;
		p[i].z += v[i].vz * dt;
	}
}

extern "C" void bodyForce_cuda(void *buffers[], void *_args)
{
	/* length of the vector */
	unsigned int n = STARPU_VECTOR_GET_NX(buffers[0]);

	/* local copy of the vector pointer */
	Pos *pos = (Pos *)STARPU_VECTOR_GET_PTR(buffers[0]);
	Vel *vel = (Vel *)STARPU_VECTOR_GET_PTR(buffers[1]);

	/* extract the value arguments */
	int offset;
	starpu_codelet_unpack_args(_args, &offset);

	unsigned threads_per_block = 64;
	unsigned nblocks = 60;

	bodyForce<<<nblocks, threads_per_block, 0, starpu_cuda_get_local_stream()>>>(pos, vel, n, offset);

	hipStreamSynchronize(starpu_cuda_get_local_stream());
}

extern "C" void integratePositions_cuda(void *buffers[], void *_args)
{
	/* length of the vector */
	unsigned int n = STARPU_VECTOR_GET_NX(buffers[0]);

	/* local copy of the vector pointer */
	Pos *pos = (Pos *)STARPU_VECTOR_GET_PTR(buffers[0]);
	Vel *vel = (Vel *)STARPU_VECTOR_GET_PTR(buffers[1]);

	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block - 1) / threads_per_block;

	integratePositions<<<nblocks, threads_per_block, 0, starpu_cuda_get_local_stream()>>>(pos, vel, n);

	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
